#include "hip/hip_runtime.h"

/*!
 * \file att_sampler.cu
 * \author Heliang Zheng
 * \adapted from https://github.com/apache/incubator-mxnet/blob/master/src/operator/bilinear_sampler.cu
*/

#include "./att_sampler-inl.h"
#include <algorithm>
#include "../../common/cuda_utils.h"


namespace mshadow {

 namespace cuda {
  template<typename DType>
  __device__ bool between(DType value, int lowerBound, int upperBound) {
   return (value >= lowerBound && value <= upperBound);
  }
  template<typename DType>
  __global__ void AttSamplerForwardKernel(const int i_c, const int i_h,
   const int i_w, const DType* data,
   const DType* grid, const int o_n,
   const int o_c, const int o_h,
   const int o_w, DType* out) {
   for (int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    index < o_n * o_c * o_h * o_w;
    index += blockDim.x * gridDim.x * gridDim.y) {
    // (n, c, h, w) is the element in out

    int w = index % o_w;
    int h = (index / o_w) % o_h;
    int c = (index / o_w / o_h) % o_c;
    int n = index / o_w / o_h / o_c;
    DType y_real;
    DType x_real;
    y_real = (*(grid + n * o_h * o_w * 2 + h * o_w + w + o_h * o_w) + 1) * (i_h - 1) / 2;
    x_real = (*(grid + n * o_h * o_w * 2 + h * o_w + w) + 1) * (i_w - 1) / 2;
    
    int top_left_y = static_cast<int>(floor(y_real));
    int top_left_x = static_cast<int>(floor(x_real));
    DType top_left_y_w = 1.0 - (y_real - top_left_y);
    DType top_left_x_w = 1.0 - (x_real - top_left_x);
    DType top_left_v = 0;
    DType top_right_v = 0;
    DType bottom_left_v = 0;
    DType bottom_right_v = 0;
    index_t out_index = n * o_c * o_h * o_w + c * o_h * o_w + h * o_w + w;
    int data_index = n * i_c * i_h * i_w + c * i_h * i_w + top_left_y * i_w + top_left_x;
    if (between(top_left_x, 0, i_w - 1) && between(top_left_y, 0, i_h - 1))
     top_left_v = *(data + data_index);
    if (between(top_left_x + 1, 0, i_w - 1) && between(top_left_y, 0, i_h - 1))
     top_right_v = *(data + data_index + 1);
    if (between(top_left_x, 0, i_w - 1) && between(top_left_y + 1, 0, i_h - 1))
     bottom_left_v = *(data + data_index + i_w);
    if (between(top_left_x + 1, 0, i_w - 1) && between(top_left_y + 1, 0, i_h - 1))
     bottom_right_v = *(data + data_index + i_w + 1);
    *(out + out_index) = top_left_v * top_left_y_w * top_left_x_w +
     top_right_v * top_left_y_w * (1.0 - top_left_x_w) +
     bottom_left_v * (1.0 - top_left_y_w) * top_left_x_w +
     bottom_right_v * (1.0 - top_left_y_w) * (1.0 - top_left_x_w);
   }
  }

  template<typename DType>
  __global__ void AttSamplerBackwardKernel(const int i_c, const int i_h,
   const int i_w, const DType* grad,
   const DType* data, const int o_n,
   const int o_c, const int o_h,
   const int o_w, DType* g_input,
   const DType* grid_src) {
   for (int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    index < o_n * o_h * o_w;
    index += blockDim.x * gridDim.x * gridDim.y) {
    // (n, c, h, w) is the element in grad
    int w = index % o_w;
    int h = (index / o_w) % o_h;
    int n = index / o_w / o_h;
    DType y_real;
    DType x_real;
    y_real = (*(grid_src + n * o_h * o_w * 2 + h * o_w + w + o_h * o_w) + 1) * (i_h - 1) / 2;
    x_real = (*(grid_src + n * o_h * o_w * 2 + h * o_w + w) + 1) * (i_w - 1) / 2;
    int top_left_y = static_cast<int>(floor(y_real));
    int top_left_x = static_cast<int>(floor(x_real));
    DType top_left_y_w = 1.0 - (y_real - top_left_y);
    DType top_left_x_w = 1.0 - (x_real - top_left_x);
    for (index_t c = 0; c < o_c; ++c) {
     index_t grad_index = n * o_c * o_h * o_w + c * o_h * o_w + h * o_w + w;
     int data_index = n * i_c * i_h * i_w + c * i_h * i_w + top_left_y * i_w + top_left_x;
     if (between(top_left_x, 0, i_w - 1) && between(top_left_y, 0, i_h - 1)) {
      atomicAdd(&g_input[data_index], *(grad + grad_index) * top_left_y_w * top_left_x_w);
     }
     if (between(top_left_x + 1, 0, i_w - 1) && between(top_left_y, 0, i_h - 1)) {
      atomicAdd(&g_input[data_index + 1], *(grad + grad_index) * top_left_y_w
       * (1.0 - top_left_x_w));
     }
     if (between(top_left_x, 0, i_w - 1) && between(top_left_y + 1, 0, i_h - 1)) {
      atomicAdd(&g_input[data_index + i_w], *(grad + grad_index) * (1.0 - top_left_y_w)
       * top_left_x_w);
     }
     if (between(top_left_x + 1, 0, i_w - 1) && between(top_left_y + 1, 0, i_h - 1)) {
      atomicAdd(&g_input[data_index + i_w + 1], *(grad + grad_index) * (1.0 - top_left_y_w)
       * (1.0 - top_left_x_w));
     }
    }
    

   }
  }
 }  // namespace cuda

 template<typename DType>
 inline void AttSamplerForward(const Tensor<gpu, 4, DType> &output,
  const Tensor<gpu, 4, DType> &input,
  const Tensor<gpu, 4, DType> &grid_src) {
  DType *out = output.dptr_;
  const DType *data = input.dptr_;
  const DType *grid = grid_src.dptr_;
  int o_n = output.size(0), o_c = output.size(1), o_h = output.size(2), o_w = output.size(3);
  int i_c = input.size(1), i_h = input.size(2), i_w = input.size(3);
  using namespace cuda;
  const int max_block = (output.shape_.Size() + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
  const int grid_dim_x = (max_block > kMaxGridDim) ? kMaxGridDim : max_block;
  const int grid_dim_y =
   (max_block > kMaxGridDim) ? (max_block + kMaxGridDim - 1) / kMaxGridDim : 1;
  dim3 num_blocks(grid_dim_x, grid_dim_y);
  dim3 threads_per_block(kMaxThreadsPerBlock);
  CheckLaunchParam(num_blocks, threads_per_block, "attention sampler forward");
  hipStream_t stream = Stream<gpu>::GetStream(output.stream_);
  cuda::AttSamplerForwardKernel<DType> << <num_blocks, threads_per_block, 0, stream >> >(
   i_c, i_h, i_w, data, grid, o_n, o_c, o_h, o_w, out);
  // post kernel check
  hipError_t err = hipPeekAtLastError();
  CHECK_EQ(err, hipSuccess) << hipGetErrorString(err);
 }

 template<typename DType>
 inline void AttSamplerBackward(const Tensor<gpu, 4, DType> &input_grad,
  const Tensor<gpu, 4, DType> &output_grad,
  const Tensor<gpu, 4, DType> &input_data,
  const Tensor<gpu, 4, DType> &grid) {
  DType *g_input = input_grad.dptr_;
  const DType *grid_src = grid.dptr_;
  const DType *grad = output_grad.dptr_;
  const DType *data = input_data.dptr_;
  int o_n = output_grad.size(0), o_c = output_grad.size(1),
   o_h = output_grad.size(2), o_w = output_grad.size(3);
  int i_c = input_data.size(1), i_h = input_data.size(2), i_w = input_data.size(3);
  using namespace cuda;
  const int max_block = (output_grad.shape_.Size() / o_c + kMaxThreadsPerBlock - 1)
   / kMaxThreadsPerBlock;
  const int grid_dim_x = (max_block > kMaxGridDim) ? kMaxGridDim : max_block;
  const int grid_dim_y =
   (max_block > kMaxGridDim) ? (max_block + kMaxGridDim - 1) / kMaxGridDim : 1;
  dim3 num_blocks(grid_dim_x, grid_dim_y);
  dim3 threads_per_block(kMaxThreadsPerBlock);
  CheckLaunchParam(num_blocks, threads_per_block, "att sampler backward");
  hipStream_t stream = Stream<gpu>::GetStream(input_grad.stream_);
  cuda::AttSamplerBackwardKernel<DType> << <num_blocks, threads_per_block, 0, stream >> >(
   i_c, i_h, i_w, grad, data, o_n, o_c, o_h, o_w, g_input, grid_src);
  //   post kernel check
  hipError_t err = hipPeekAtLastError();
  CHECK_EQ(err, hipSuccess) << hipGetErrorString(err);
 }

}  // namespace mshadow



namespace mxnet {
 namespace op {
  template<>
  Operator* CreateOp<gpu>(AttSamplerParam param, int dtype) {
   Operator *op = NULL;
   MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new AttSamplerOp<gpu, DType>(param);
   })
    return op;
  }

 }  // namespace op
}  // namespace mxnet
